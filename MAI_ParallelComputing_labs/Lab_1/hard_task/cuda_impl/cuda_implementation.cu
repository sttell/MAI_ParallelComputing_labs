#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include "cuda_wrappers.hpp"
#include "../common/CUDA_common.hpp"
#include "../common/GLOB_common.hpp"

#define MAX_THREADS_PER_BLOCK 1024
#define BLOCK_SIZE 32

/**
 * @brief Calculate sqrt for all element in array.
 *
 * @param data - float array with number
 */
__global__ void cuda_sqrt(float* dst, float* mat1, float* mat2, int cols) {
    int idx = (blockIdx.y * blockDim.y + threadIdx.y) * cols + blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = mat1[idx] + mat2[idx];    
}


/**
 * @brief Calculates the square root of each element of the array using the CUDA kernel
 * 
 * @param array           - Array with values
 * @param num_of_elements - Number of elements in the array
 * @param out_mode        - Output mode of results
 * @return int - Net operating time of CUDA kernel in microseconds
 */
int CUDA_sqrt_array(float* dst, float* mat1, float* mat2, int N, int M, OutMode out_mode) {
    
    if (BLOCK_SIZE * BLOCK_SIZE > MAX_THREADS_PER_BLOCK) {
        throw std::runtime_error("Block size is soo big. Expected size: 1024 threads.");
    }

    if (N < 1 || M < 1) 
        throw std::runtime_error("Number of array elements must be only > 1.");
    
    int sizeof_matrix = N * M * sizeof(float);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, M / threads.y);

    hipEvent_t start, stop;
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    // Memory allocate
    float* out_dev;
    float* in1_dev;
    float* in2_dev;
    {
        checkCudaErrors(hipMalloc(&in1_dev, sizeof_matrix));
        checkCudaErrors(hipMemcpy(in1_dev, mat1, sizeof_matrix, hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc(&in2_dev, sizeof_matrix));
        checkCudaErrors(hipMemcpy(in2_dev, mat2, sizeof_matrix, hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc(&out_dev, sizeof_matrix));
    }

    if (out_mode == OutMode::FormattedOutput) {
        std::cout << "Matrix size:  {" << N << ", " << M << "}\n";
        std::cout << "CUDA threads: {" << threads.x << ", " << threads.y << "}\n";
        std::cout << "CUDA blocks : {" << blocks.x  << ", " << blocks.y << "}\n";
    }

    // Calculation
    {
        hipEventRecord(start);
        cuda_sqrt<<<blocks, threads>>>(out_dev, in1_dev, in2_dev, M);
        
        checkCudaErrors(hipGetLastError());

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipDeviceSynchronize(); 
        

    }
    
    // Memory deallocate
    {
        checkCudaErrors(hipMemcpy(dst, out_dev, sizeof_matrix, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(out_dev));
        checkCudaErrors(hipFree(in1_dev));
        checkCudaErrors(hipFree(in2_dev));
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    int microseconds = static_cast<int>(1000.f * milliseconds);

    switch (out_mode) {
        case OutMode::FormattedOutput:
        {
            std::cout << "CUDA time simple (microseconds): " << microseconds << std::endl;
            global_utils::save_array_to_stdout(dst, N * M);
            break;
        }
        case OutMode::TableOutput:
        {
            std::cout << N * M << ";" << microseconds << std::endl;
            break;
        }
    }

    return microseconds;
}
