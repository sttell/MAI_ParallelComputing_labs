#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include "cuda_wrappers.hpp"
#include "../common/CUDA_common.hpp"
#include "../common/GLOB_common.hpp"

#define THREAD_NUM 1024


/**
 * @brief Calculate sqrt for all element in array.
 *
 * @param data - float array with number
 */
__global__ void cuda_sqrt(float* data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[idx] = sqrt(data[idx]);
}


/**
 * @brief Calculates the square root of each element of the array using the CUDA kernel
 * 
 * @param array           - Array with values
 * @param num_of_elements - Number of elements in the array
 * @param out_mode        - Output mode of results
 * @return int - Net operating time of CUDA kernel in microseconds
 */
int CUDA_sqrt_array(float* array, int num_of_elements, OutMode out_mode) {
    
    if (num_of_elements < 1) 
        throw std::runtime_error("Number of array elements must be only > 1.");
    
    int num_bytes = num_of_elements * sizeof(float);

    dim3 threads(THREAD_NUM);
    dim3 blocks(num_of_elements / THREAD_NUM);

    hipEvent_t start, stop;
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    // Memory allocate
    float* in_dev;
    {
        checkCudaErrors(hipMalloc(&in_dev, num_bytes));
        checkCudaErrors(hipMemcpy(in_dev, array, num_bytes, hipMemcpyHostToDevice));
    }

    // Calculation
    {
        hipEventRecord(start);
        cuda_sqrt<<<blocks, threads>>>(in_dev);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipDeviceSynchronize(); 
        checkCudaErrors(hipGetLastError());

    }
    
    // Memory deallocate
    {
        checkCudaErrors(hipMemcpy(array, in_dev, num_bytes, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(in_dev));
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    int microseconds = static_cast<int>(1000.f * milliseconds);

    switch (out_mode) {
        case OutMode::FormattedOutput:
        {
            std::cout << "CUDA time simple (microseconds): " << microseconds << std::endl;
            global_utils::save_array_to_stdout(array, num_of_elements);
            break;
        }
        case OutMode::TableOutput:
        {
            std::cout << num_of_elements << ";" << microseconds << std::endl;
            break;
        }
    }

    return microseconds;
}
